
#include <hip/hip_runtime.h>

__global__ void sumKernel( float *a, float *b, float *c )
{
	int idx = threadIdx.x + blockIdx.x*blockDim.x;
	c[idx] = a[idx] + b[idx];
}