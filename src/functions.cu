#include "hip/hip_runtime.h"
#include "Opener.h"

#include <iostream>
using std::cout;
using std::endl;

extern __global__ void sumKernel( float *a, float *b, float *c );

void hnd(const hipError_t &err) {
	if(err != hipSuccess) 
		cout<<"hipError_t = "<<hipGetErrorString(err)<<endl;
}

void funcCuda( float *v1, int N1, float *v2, int N2, float* out, int N3 )
{
	typedef Opener<float, hipMalloc, hipFree> FloatOpener;
	
	Timer timer;
	timer.start();

	int N = N1 < N2 ? N1 : N2;

	float *v1Dev = NULL;
	float *v2Dev = NULL;
	float *outDev = NULL;

	FloatOpener op1(v1Dev,  N);
	FloatOpener op2(v2Dev,  N);
	FloatOpener op3(outDev, N);
	
	int numBytes = N*sizeof(float);
	
	hnd( hipMemcpy( v1Dev, v1, numBytes, hipMemcpyHostToDevice ) );
	hnd( hipMemcpy( v2Dev, v2, numBytes, hipMemcpyHostToDevice ) );
	
	dim3 threads = dim3(64, 1);
	dim3 blocks = dim3(N/threads.x, 1);

	sumKernel<<<blocks, threads>>>( v1Dev, v2Dev, outDev );

	hnd( hipMemcpy( out, outDev, numBytes, hipMemcpyDeviceToHost ) );

	timer.stop();
	cout<<"gpuTime = "<<timer.elapsed()<<endl;
}