#include "hip/hip_runtime.h"
#include "Opener.h"

#include <iostream>
using std::cout;
using std::endl;

extern __global__ void sumKernel( float *a, float *b, float *c );

void hnd(const hipError_t &err) {
	if(err != hipSuccess) 
		cout<<"hipError_t = "<<hipGetErrorString(err)<<endl;
}

void print(const hipDeviceProp_t &prop) 
{
	cout<<"\n###Device Properties###"<<endl;
	
	cout<<"Name: "<<prop.name<<endl;
	cout<<"Capability: "<<prop.major<<"."<<prop.minor<<endl;
	cout<<"Clock Rate: "<<prop.clockRate<<endl;
	cout<<"Overlap: "<<prop.deviceOverlap<<endl;
	cout<<"Timeout Enabled: "<<prop.kernelExecTimeoutEnabled<<endl;

	cout<<"Total Global Memory: "<<prop.totalGlobalMem<<endl;
	cout<<"Total Constant Memory: "<<prop.totalConstMem<<endl;
	cout<<"Max Memory Pitch: "<<prop.memPitch<<endl;
	cout<<"Texture Alignment: "<<prop.textureAlignment<<endl;

	cout<<"MultiProcessor Count: "<<prop.multiProcessorCount<<endl;
	cout<<"Shared Memory per MP: "<<prop.sharedMemPerBlock<<endl;
	cout<<"Registers per Block: "<<prop.regsPerBlock<<endl;
	cout<<"Warp Size: "<<prop.warpSize<<endl;
	cout<<"Max Threads per Block: "<<prop.maxThreadsPerBlock<<endl;

	cout<<"Max Threads Dim ("<<prop.maxThreadsDim[0]<<", "<<prop.maxThreadsDim[1]<<", "<<prop.maxThreadsDim[2]<<")"<<endl;
	cout<<"Max Grid Size ("<<prop.maxGridSize[0]<<", "<<prop.maxGridSize[1]<<", "<<prop.maxGridSize[2]<<")"<<endl;
}

void properties() 
{
	cout<<"properties"<<endl;
	
	int count;
	hnd( hipGetDeviceCount( &count ) );
	
	hipDeviceProp_t prop;

	for (int i = 0; i < count; ++i) 
	{
		hnd( hipGetDeviceProperties( &prop, i ) );
		print(prop);
	}

}

void funcCuda( float *v1, int N1, float *v2, int N2, float* out, int N3 )
{
	typedef Opener<float, hipMalloc, hipFree> FloatOpener;
	properties();
	Timer timer;
	timer.start();

	int N = N1 < N2 ? N1 : N2;

	float *v1Dev = NULL;
	float *v2Dev = NULL;
	float *outDev = NULL;

	FloatOpener op1(v1Dev,  N);
	FloatOpener op2(v2Dev,  N);
	FloatOpener op3(outDev, N);
	
	int numBytes = N*sizeof(float);
	
	hnd( hipMemcpy( v1Dev, v1, numBytes, hipMemcpyHostToDevice ) );
	hnd( hipMemcpy( v2Dev, v2, numBytes, hipMemcpyHostToDevice ) );
	
	dim3 threads = dim3(64, 1);
	dim3 blocks = dim3(N/threads.x, 1);

	sumKernel<<<blocks, threads>>>( v1Dev, v2Dev, outDev );

	hnd( hipMemcpy( out, outDev, numBytes, hipMemcpyDeviceToHost ) );

	timer.stop();
	cout<<"gpuTime = "<<timer.elapsed()<<endl;
}
